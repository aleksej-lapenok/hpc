#include <hip/hip_runtime.h>

#include <iostream>
#include <memory>
#include <string>


#include <stdio.h>




#ifndef BLOCK_SIZE
# define BLOCK_SIZE 16
#endif

#ifndef _M
# define _M 10000
#endif

#ifndef _N
# define _N 10000
#endif

#if !defined(CUDA) && !defined(CPU) && !defined(CHECK)
# define CUDA
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"gpuAssert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void mx_dist(float *m_in, float *m_out, int m, int n) 
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; 
    int j = blockIdx.x * blockDim.x + threadIdx.x;
	float s = 0, sum = 0;

    if( i < m && j < m) {

    	for(int k = 0; k < n; ++k) {
    		s = m_in[i*m + k] - m_in[j*m + k];
    		sum += s*s;
    	}

    	// printf("--> %d %d %f %f\n", j, i, m_in[j*n], sum);
    	m_out[i*m + j] = sum;
    }
}

void mx_dist_cpu(float *m_in, float *m_out, int m, int n) 
{ 
	float s, sum;
    
	for(int i = 0; i < m; ++i) 
		for(int j = 0; j < m; ++j) {
			sum = 0;
			for(int k = 0; k < n; ++k) {
				s = m_in[i*m + k] - m_in[j*m + k];
				sum += s*s;
			}
			m_out[i*m + j] = sum;
		}
}

void init_mx(float *A, size_t m, size_t n) 
{
	for(int i = 0; i < m; ++i) {		
		for(int j = 0; j < n; ++j) {
			float t = sin(i*m + j) * 10 + 1; 
			A[i*m + j] = t;
		}
	}
}
void print_mx(float *A, size_t m, size_t n) 
{
	for(int i = 0; i < m; ++i) {		
		for(int j = 0; j < n; ++j) {
			printf("%d %d %f\n", i, j, A[i*m + j]);			
		}
	}
}

void cmp_mx(float *A, float *B, size_t m, size_t n) 
{
	for(int i = 0; i < m; ++i) {		
		for(int j = 0; j < n; ++j) {
			if( abs(A[i*m + j] - B[i*m + j]) > 0.01) {
				printf("not equal %f %f\n", A[i*m + j], B[i*m + j]);
				return;
			} else {
				printf("Equal\n");
			}
		}
	}
}



float *run_cuda(float *A, size_t m, size_t n) 
{
	hipError_t e;

	float *A_d;
	float *B, *B_d;

	B = (float*) malloc(m*m*sizeof(float));


	e = hipMalloc(&A_d, m*n*sizeof(float));
	gpuErrchk(e);
	e = hipMalloc(&B_d, m*m*sizeof(float));
	gpuErrchk(e);


	e = hipMemcpy(A_d, A, m*n*sizeof(float), 
				hipMemcpyHostToDevice);
	gpuErrchk(e);	


    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	mx_dist<<<dimGrid, dimBlock>>>(A_d, B_d, m, n);


	e = hipMemcpy(B, B_d, m*m*sizeof(float), 
				hipMemcpyDeviceToHost);
	gpuErrchk(e);


	hipFree(A_d);
	hipFree(B_d);

	
	return B;
}


float *run_cpu(float *A, size_t m, size_t n) 
{    
	
	float *B;
	B = (float*) malloc(m*m*sizeof(float));

	mx_dist_cpu(A, B, m, n);

	return B;
}

int main() 
{

	int m = _M, n = _N;
	float *A;
	A = (float*) malloc(m*n*sizeof(float));
	init_mx(A, m, n);

#if defined(CUDA) | defined(CHECK)
	float *gpu = run_cuda(A, m, n);
#endif

#if defined(CPU) | defined(CHECK)
	float *cpu = run_cpu(A, m, n);
#endif

#if defined(CHECK)
	cmp_mx(gpu, cpu, m, m);
#endif
	//for(int _j = 0; _j < size; ++_j) printf("%f ", h_vec[2][_j]);
	// printf("\n");

    
    return 0;
}